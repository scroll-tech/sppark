#include <hip/hip_runtime.h>
#include <cstdio>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377.hpp>
#elif defined(FEATURE_BN254)
# include <ff/alt_bn128.hpp>
#else
# error "no FEATURE"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>
typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;
#include <msm/pippenger.cuh> //needs previous typedef


struct FFITraitObject{
    uint64_t *ptr;
};
typedef FFITraitObject Point;
typedef FFITraitObject Affine;
typedef FFITraitObject Scalar;


#ifndef __CUDA_ARCH__
extern "C"
void benchmark_scalar_mul_sppark(const Scalar *scalar, uint64_t loop_times, uint64_t repeat_times) {

    scalar_t *d_scalar;
    hipMalloc(&d_scalar, sizeof(scalar_t));
    hipMemcpy((uint64_t*)d_scalar, scalar->ptr, sizeof(scalar_t), hipMemcpyHostToDevice);
    for(uint64_t i = 0; i < repeat_times; i++){
        op_scalar_mul_sppark<<<1, 1>>>(d_scalar, loop_times);
    }
    hipFree(d_scalar);
}
#endif


#ifndef __CUDA_ARCH__
extern "C"
void benchmark_point_mixed_add_sppark(const Point *point, uint64_t loop_times, uint64_t repeat_times) {

    const size_t ELT_LIMBS = 4 * sizeof(uint64_t); // FF : 256bit = 32bytes
    const size_t AFF_POINT_LIMBS = 2 * ELT_LIMBS;  // X Y
    const size_t JAC_POINT_LIMBS = 4 * ELT_LIMBS;  // X Y Z

    affine_t *d_point;
    bucket_t *d_res;
    hipMalloc(&d_point, JAC_POINT_LIMBS);
    hipMalloc(&d_res, sizeof(bucket_t));
    hipMemcpy((uint64_t*)d_point, point->ptr, AFF_POINT_LIMBS, hipMemcpyHostToDevice);
    for(uint64_t i = 0; i < repeat_times; i++){
        op_point_mixed_add_sppark<<<1, 1>>>(d_point, loop_times, d_res);
    }
    hipFree(d_point);
    hipFree(d_res);
}
#endif
