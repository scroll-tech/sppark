// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377.hpp>
#elif defined(FEATURE_BN254)
# include <ff/alt_bn128.hpp>
#else
# error "no FEATURE"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

struct FFITraitObject{
    uint64_t *ptr;
};
typedef FFITraitObject Point;
typedef FFITraitObject Affine;
typedef FFITraitObject Scalar;

#ifndef __CUDA_ARCH__
extern "C"
RustError mult_pippenger_halo2(
    const Point *out,
    const Affine *point,
    uint64_t npoints,
    const Scalar *scalar,
    size_t ffi_affine_sz,
    uint64_t gpu_idx)
{
    hipSetDevice(gpu_idx);
    point_t* ret = (point_t*)out->ptr;
    const affine_t* points = (affine_t*)(point->ptr);
    const scalar_t* scalars = (scalar_t*)(scalar->ptr);
    return mult_pippenger<bucket_t>(ret, points, npoints, scalars, true);
}
#endif
